
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <iostream>
#include <chrono>

using namespace std;

float makeCPU(int* inData, int N);

float cudaParallel(int* inData, int N);

void init(int* inData, int N)
{
	for (int i = 0; i < N; i++)
		inData[i] = 100 - i + 1;
}

__global__ void reductionKernelMinimum(int *inData, int N)
{
	int tId = threadIdx.x;
	int k = blockIdx.x * blockDim.x + threadIdx.x;

	int before = k;
	int after = (k + 1);

	while (after < N)
	{
		if (inData[before] < inData[after])
			continue;
		else
			inData[before] = inData[after];

		before *= 2;
		after *= 2;

		if (before >= N)
			break;

		__syncthreads();
	}
}

int main()
{
	int N;
	while (true)
	{
		cout << "Enter number of elements: " << endl;
		cin >> N;
		const int elementsCount = N;

		cout << "Reduction for: " << elementsCount << endl;

		int *a = new int[elementsCount];

		init(a, elementsCount);
		float gpuTime = cudaParallel(a, elementsCount);
		cout << "Time on gpu: " << gpuTime << endl;
		
		init(a, elementsCount);
		float cpuTime = makeCPU(a, elementsCount);
		cout << "Time on cpu in ns " << cpuTime << endl;
 	}

	return 0;
}

float makeCPU(int* inData, int N)
{
	int min = inData[0];

	std::chrono::time_point<std::chrono::system_clock> start, end;
	start = std::chrono::system_clock::now();
	for (int i = 1; i < N; i++) 
	{
		if (inData[i] < min) 
			min = inData[i];
	}
	end = std::chrono::system_clock::now();

	int elapsed = std::chrono::duration_cast<std::chrono::nanoseconds>
		(end - start).count();

	return elapsed;
}

float cudaParallel(int* inData, int N)
{
	int* deviceData;

	hipMalloc((void**)&deviceData, N * sizeof(int));
	hipMemcpy(deviceData, inData, N * sizeof(int), hipMemcpyHostToDevice);

	hipEvent_t start, stop;
	float gpuTime = 0.0f;

	hipEventCreate(&start);
	hipEventCreate(&stop);


	dim3 threads(256, 1, 1);
	dim3 blocks(N / 256, 1);

	hipEventRecord(start, 0);

	reductionKernelMinimum <<<blocks, threads>>> (deviceData, N);

	hipEventRecord(stop, 0);

	hipEventSynchronize(stop);
	hipEventElapsedTime(&gpuTime, start, stop);

	hipEventDestroy(start);
	hipEventDestroy(stop);

	hipMemcpy(inData, deviceData, N * sizeof(int), hipMemcpyDeviceToHost);

	hipFree(deviceData);

	return gpuTime;
}